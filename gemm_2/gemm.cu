#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <float.h>
#include "utils.h"

using T = cute::half_t;
using namespace cute;

template <typename T, int BM, int BN, int BK, typename TiledMMA, 
            typename G2SCopyA, typename G2SCopyB,
            typename SmemLayoutA, typename SmemLayoutB, 
            typename S2RCopyAtomA, typename S2RCopyAtomB>
__global__ void gemm_shm_v2(const T *Aptr, const T *Bptr, T *Cptr, int m, int n, int k) {
    // Initilize thread block
    int idx = threadIdx.x;
    int ix = blockIdx.x;
    int iy = blockIdx.y;

    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor C = make_tensor(make_gmem_ptr(Cptr), make_shape(m, n), make_stride(n, Int<1>{}));

    // Global Memory
    Tensor gA = local_tile(A, make_tile(Int<BM>{}, Int<BK>{}), make_coord(iy, _)); // (BM, BK, num_tile_k)
    Tensor gB = local_tile(B, make_tile(Int<BN>{}, Int<BK>{}), make_coord(ix, _)); // (BN, BK, num_tile_k)
    Tensor gC = local_tile(C, make_tile(Int<BM>{}, Int<BN>{}), make_coord(iy, ix)); // (BM, BN) 


    // Initilize shared memory
    extern __shared__ T shm_data[];
    T *Ashm = shm_data;
    T *Bshm = shm_data + cute::cosize(SmemLayoutA{});
    auto sA = make_tensor(make_smem_ptr(Ashm),SmemLayoutA{}); // (BM, BK)
    auto sB = make_tensor(make_smem_ptr(Bshm), SmemLayoutB{}); // (BN, BK)

    // from global memory to shared memory
    G2SCopyA g2s_tiled_copy_a;
    auto g2s_thr_copy_a = g2s_tiled_copy_a.get_slice(idx);
    auto tAgA_copy = g2s_thr_copy_a.partition_S(gA); // (CPY, CPY_M, CPY_K, k)
    auto tAsA_copy = g2s_thr_copy_a.partition_D(sA); // (CPY, CPY_M, CPY_K)

    G2SCopyB g2s_tiled_copy_b;
    auto g2s_thr_copy_b = g2s_tiled_copy_b.get_slice(idx);
    auto tBgB_copy = g2s_thr_copy_b.partition_S(gB); // (CPY, CPY_N, CPY_K, k)
    auto tBsB_copy = g2s_thr_copy_b.partition_D(sB); // (CPY, CPY_N, CPY_K)


    // register, use tiled_mma to partition register A/B/C
    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
    auto tCgC = thr_mma.partition_C(gC); // (MMA, MMA_M, MMA_N)

    auto tCrA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
    auto tCrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
    auto tCrC = thr_mma.partition_fragment_C(gC);           // (MMA, MMA_M, MMA_N)
    clear(tCrC);


    // from shared memory to register, use tiled_mma to generate tiled_copy
    auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, tiled_mma);
    auto s2r_thr_copy_a = s2r_tiled_copy_a.get_slice(idx);
    auto tCsA = s2r_thr_copy_a.partition_S(sA);     // (CPY, CPY_M, CPY_K)
    auto tCrA_view = s2r_thr_copy_a.retile_D(tCrA); // (CPY, CPY_M, CPY_K)

    auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, tiled_mma);
    auto s2r_thr_copy_b = s2r_tiled_copy_b.get_slice(idx);
    auto tCsB = s2r_thr_copy_b.partition_S(sB);     // (CPY, CPY_N, CPY_K)
    auto tCrB_view = s2r_thr_copy_b.retile_D(tCrB); // (CPY, CPY_N, CPY_K)


//   if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0)
//   {
//       PRINT("tCsA", tCsA.shape())     
//       PRINT("tCrA_view", tCrA_view.shape()) 

//       PRINT("tCsB", tCsB.shape())     
//       PRINT("tCrB_view", tCrB_view.shape()) 
//   }

  // loop over k: i. load tile, ii. mma
  int ntile = k / BK;
#pragma unroll 1
  for (int itile = 0; itile < ntile; ++itile)
  {
    // copy  (CPY, CPY_M, CPY_K) , async
    cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, itile),
               tAsA_copy(_, _, _));
    cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, itile),
               tBsB_copy(_, _, _));

    cp_async_fence();
    cp_async_wait<0>();
    __syncthreads();
    
    cute::copy(s2r_tiled_copy_a, tCsA, tCrA_view);
    cute::copy(s2r_tiled_copy_b, tCsB, tCrB_view);
    cute::gemm(tiled_mma, tCrC, tCrA, tCrB, tCrC);
  } // itile

  // register to global memory
  cute::copy(tCrC, tCgC);
}

template <typename T>
void gemm_v2(const T *a, const T *b, T *c, int M, int N, int K) {

    auto BM = Int<128>{};
    auto BN = Int<256>{};
    auto BK = Int< 32>{};
    // Define the smem layouts
    using SmemLayoutAtom = decltype(composition(
        Swizzle<3, 3, 3>{},
        make_layout(make_shape(Int<8>{}, Int<BK>{}),
                    make_stride(Int<BK>{}, Int<1>{}))));
    using SmemLayoutA = decltype(tile_to_shape(SmemLayoutAtom{},
                                               make_shape(Int<BM>{}, Int<BK>{})));
    using SmemLayoutB = decltype(tile_to_shape(SmemLayoutAtom{},
                                               make_shape(Int<BN>{}, Int<BK>{})));                    // (m,n) -> smem_idx

    // mma
    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    static constexpr int kMmaEURepeatM = 2;
    static constexpr int kMmaEURepeatN = 2;
    static constexpr int kMmaEURepeatK = 1;

    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int kMmaPM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
    static constexpr int kMmaPN = 2 * kMmaEURepeatN * get<1>(mma_atom_shape{});
    static constexpr int kMmaPK = 1 * kMmaEURepeatK * get<2>(mma_atom_shape{});
    using MMA_EU_RepeatT = decltype(make_layout(make_shape(
        Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
    using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
  
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));

    // copy from global memory to shared memory
    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, T>;
    using G2SCopyA =
        decltype(make_tiled_copy(g2s_copy_atom{},
                                 make_layout(make_shape(Int<32>{}, Int<4>{}), // Thr layout 32x4 k-major
                                             make_stride(Int<4>{}, Int<1>{})),
                                 make_layout(make_shape(Int<1>{}, Int<8>{})))); // Val layout 1x8
    using G2SCopyB = G2SCopyA;

    // copy from shared memory to register
    // use mma tiled ,so no tiled here
    using s2r_copy_op = SM75_U32x4_LDSM_N;
    using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
    using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;
    using S2RCopyAtomA = s2r_copy_atom;
    using S2RCopyAtomB = s2r_copy_atom;

    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    dim3 block(size(MMA{}));
    dim3 grid(BX, BY);

    // C_shm is shared with A_shm and B_shm
    static constexpr int shm_size_AB =
        cute::cosize(SmemLayoutA{}) + cute::cosize(SmemLayoutB{});
    static constexpr int kShmSize =
        shm_size_AB * sizeof(T);

    int shm_size = kShmSize;

    hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_shm_v2<T), BM, BN, BK, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, S2RCopyAtomA, S2RCopyAtomB>,
                         hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    
    gemm_shm_v2<T, BM, BN, BK, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, S2RCopyAtomA, S2RCopyAtomB>
               <<<grid, block, shm_size>>>(a, b, c, M, N, K);
}

int main() {

    const int test_num = 64;
    int M_list[test_num];
    int N_list[test_num];
    int K_list[test_num];

    for (int i = 0; i < test_num; i++) {
        M_list[i] = (i + 1) * 256;
        N_list[i] = (i + 1) * 256;
        K_list[i] = (i + 1) * 256;
    }

    const int outer_repeat = 10, inner_repeat = 1;

    printf("\nalgo = Cute_HGEMM_V2\n");
    for (int j = 0; j < 5; j++) {
        int M = M_list[j], N = N_list[j], K = K_list[j];
        float max_error = testF16F16GemmMaxError_V2<T>(gemm_v2, M, N, K);
        printf("M N K = %6d %6d %6d, ", M, N, K);
        printf("Max Error = %f\n", max_error);
    }

    for (int j = 0; j < test_num; j++) {
        int M = M_list[j], N = N_list[j], K = K_list[j];
 
        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int k = 0; k < outer_repeat; k++) {
            double this_sec = testF16F16GemmPerformance<T>(
                gemm_v2, M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

        printf("M N K = %6d %6d %6d, ", M, N, K);
        printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
        printf("AVG Performance = %10.4lf Gflops\n", avg_Gflops);
    }


    return 0;
}